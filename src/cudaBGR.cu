#include "hip/hip_runtime.h"
#include "cudaBGR.h"

//-------------------------------------------------------------------------------------------------------------------------

template<bool isBGR>
__global__ void RGBAToRGBAf(uchar4* srcImage,
                           float4* dstImage,
                           int width, int height)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	const int pixel = y * width + x;

	if( x >= width )
		return; 

	if( y >= height )
		return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);
		
	const float  s  = 1.0f;
	const uchar4 px = srcImage[pixel];
	
	if( isBGR )
		dstImage[pixel] = make_float4(px.z * s, px.y * s, px.x * s, px.w * s);
	else
		dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, px.w * s);
}

/**
 * Convert 8-bit fixed-point BGRA image to 32-bit floating-point RGBA image
 * @ingroup util
 */
hipError_t cudaBGRA8ToRGBA32( uchar4* srcDev, float4* destDev, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	const dim3 blockDim(8,8,1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

	RGBAToRGBAf<true><<<gridDim, blockDim>>>( srcDev, destDev, width, height );
	
	return CUDA(hipGetLastError());
}

